#include "hip/hip_runtime.h"
#include "ptychofft.cuh"
#include "kernels.cu"

// constructor, memory allocation
ptychofft::ptychofft(size_t ptheta, size_t nz, size_t n, size_t nscan,
  size_t ndet, size_t nprb
) :
  ptheta(ptheta), nz(nz), n(n), nscan(nscan), ndet(ndet),
  nprb(nprb)
{
	// create batched 2D FFT plan on GPU with sizes (ndet, ndet)
  // transform shape MUST be less than or equal to input and ouput shapes.
	int ffts[2] = {(int)ndet, (int)ndet};
	hipfftPlanMany(&plan2d, 2,
    ffts,                 // transform shape
    ffts, 1, ndet * ndet, // input shape
    ffts, 1, ndet * ndet, // output shape
    HIPFFT_C2C,
    ptheta * nscan        // Number of FFTs to do simultaneously
  );
  // create a place to put the FFT and IFFT output.
  hipMalloc((void**)&fft_out, ptheta * nscan * ndet * ndet * sizeof(float2));

	// init 3d thread block on GPU
	BS3d.x = 32;
	BS3d.y = 32;
	BS3d.z = 1;

	// init 3d thread grids	on GPU
	GS3d0.x = ceil(nprb * nprb / (float)BS3d.x);
	GS3d0.y = ceil(nscan / (float)BS3d.y);
	GS3d0.z = ceil(ptheta / (float)BS3d.z);

	GS3d1.x = ceil(ndet * ndet / (float)BS3d.x);
	GS3d1.y = ceil(nscan / (float)BS3d.y);
	GS3d1.z = ceil(ptheta / (float)BS3d.z);

	GS3d2.x = ceil(nscan / (float)BS3d.x);
	GS3d2.y = ceil(ptheta / (float)BS3d.y);
	GS3d2.z = 1;
}

// destructor, memory deallocation
ptychofft::~ptychofft()
{
  free();
}

void ptychofft::free()
{
  if(!is_free)
  {
    hipfftDestroy(plan2d);
    hipFree(fft_out);
    is_free = true;
  }
}

// forward ptychography operator g = FQf
void ptychofft::fwd(size_t g_, size_t f_, size_t scan_, size_t prb_)
{
  // convert pointers to correct type
  f = (float2 *)f_;
  g = (float2 *)g_;
  scan = (float2 *)scan_;
  prb = (float2 *)prb_;

	// probe multiplication of the object array
  hipMemset(fft_out, 0, ptheta * nscan * ndet * ndet * sizeof(float2));
	muloperator<<<GS3d0, BS3d>>>(f, fft_out, prb, scan, ptheta, nz, n, nscan, nprb, ndet, 2); //flg==2 forward transform
	// Fourier transform
	hipfftExecC2C(plan2d, (hipfftComplex *)fft_out, (hipfftComplex *)g, HIPFFT_FORWARD);
}

// adjoint ptychography operator with respect to object (flg==0) f = Q*F*g, or probe (flg==1) prb = Q*F*g
void ptychofft::adj(size_t f_, size_t g_, size_t scan_, size_t prb_, int flg)
{
  // convert pointers to correct type
  f = (float2 *)f_;
  g = (float2 *)g_;
  scan = (float2 *)scan_;
  prb = (float2 *)prb_;

	// inverse Fourier transform
	hipfftExecC2C(plan2d, (hipfftComplex *)g, (hipfftComplex *)fft_out, HIPFFT_BACKWARD);
	// adjoint probe (flg==0) or object (flg=1) multiplication operator
	muloperator<<<GS3d0, BS3d>>>(f, fft_out, prb, scan, ptheta, nz, n, nscan, nprb, ndet, flg);
}
